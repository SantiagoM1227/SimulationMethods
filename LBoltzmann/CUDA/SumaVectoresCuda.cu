
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>

using namespace std;

#define Lx 16
#define Nx 8

const int Mx = (Lx+Nx-1)/Nx;

//----------Programa del Device------------------
//-----------------KERNELS-----------------------
__global__ void AddTwoVectors(float * d_a, float * d_b, float * d_c){
  //¿Qué tarea me toca?
  int ix; ix = blockIdx.x*blockDim.x+threadIdx.x;
  d_c[ix]=d_a[ix]+d_b[ix];
}
//------------Código de Host---------------------
int main(void){
  int ix;
  //Declarar todas las variables-----------------
  float h_a[Lx],h_b[Lx],h_c[Lx];
  //----------En el device-----------------------
  float *d_a; hipMalloc((void**)&d_a, Lx*sizeof(float));
  float *d_b; hipMalloc((void**)&d_b, Lx*sizeof(float));
  float *d_c; hipMalloc((void**)&d_c, Lx*sizeof(float));
  
  //Inicializar variables------------------------
  
  for (ix = 0; ix<Lx; ix++){
    h_a[ix]=ix; h_b[ix]=2*ix;
  }
  
  //Enviar al Device
  hipMemcpy(d_a,h_a,Lx*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b,Lx*sizeof(float),hipMemcpyHostToDevice);
  
  //Correr en el Device
  dim3 ThreadsPerBlock(Nx,1,1);
  dim3 BlocksPerGrid(Mx,1,1);
  
  AddTwoVectors<<<BlocksPerGrid,ThreadsPerBlock>>>(d_a,d_b,d_c);
  
  //Devolver el resultado al Host
  
  hipMemcpy(h_c,d_c,Lx*sizeof(float),hipMemcpyDeviceToHost);
  //Imprimir los resultados
  
  for (ix = 0; ix<Lx; ix++){
    cout<<h_c[ix]<<endl;
  }
  
  
  return 0;
}
